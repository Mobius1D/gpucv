#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void cu_sobel(int *orig, int *cpu, int height,int width){

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    float dx, dy;
    if( x > 0 && y > 0 && x < width-1 && y < height-1) {
        dx = (-1* orig[(y-1)*width + (x-1)]) + (-2*orig[y*width+(x-1)]) + (-1*orig[(y+1)*width+(x-1)]) +
             (    orig[(y-1)*width + (x+1)]) + ( 2*orig[y*width+(x+1)]) + (   orig[(y+1)*width+(x+1)]);
        dy = (    orig[(y-1)*width + (x-1)]) + ( 2*orig[(y-1)*width+x]) + (   orig[(y-1)*width+(x+1)]) +
             (-1* orig[(y+1)*width + (x-1)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+1)]);
        cpu[(y)*(width) + (x)] = sqrt( (dx*dx) + (dy*dy) );
    }
}

// Called from driver program.  Handles running GPU calculation
extern "C" void gpu_sobel(int *l_source_array, int *l_result_array,int src_rows, int src_column_size) {
  int num_bytes_source = src_column_size * src_rows * sizeof(int);
  int *l_source_array_d;
  int *l_result_array_d;

  hipMalloc((void **)&l_source_array_d, num_bytes_source);
  hipMemcpy(l_source_array_d, l_source_array, num_bytes_source,hipMemcpyHostToDevice);

  int result_column_size = src_column_size; 
  int result_row_size = src_rows;
  int num_bytes_result = result_column_size * result_row_size * sizeof(int);
  hipMalloc((void **)&l_result_array_d, num_bytes_result);



  dim3 threadsPerBlock(32, 32);

  dim3 numBlocks(ceil(src_column_size/32), ceil(src_rows/32), 1);

  cu_sobel<<<numBlocks, threadsPerBlock>>>(l_source_array_d, l_result_array_d,
                                      src_rows, src_column_size);

  hipMemcpy(l_result_array, l_result_array_d, num_bytes_result,
             hipMemcpyDeviceToHost);


  hipFree(l_source_array_d);
  hipFree(l_result_array_d);
}
